// Author: Jose F. Martinez Rivera
// Course: ICOM4036- 040
// Professor: Dr. Wilson Rivera
// Hand-In Date: April 29, 2013


// Problem 1:
// Assume that you start with two arrays of random numbers (float precision) 
// between -10 and 10. You are supposed to add these two arrays of length N
// on the GPU and eventually get the result into a host array of length N
// You will have to run a loop to investigate what happens when the value of N
// increases by factors of 2 from N=2^10 to N=2^20. Specifically, report in a plot 
// the amount of time (inclusive and exclusive) it takes for the program to add 
// the two arrays as a function of the array size. 
// You will have to do the sequence of steps above twice (plot all the result on 
// the same plot though). The first time, the number of threads in a block is 
// going to be 32. As the value of N increases, you’ll have to adjust the number 
// of blocks you launch to get the job done. The second time around, you are 
// going to use 1024 threads in a block. Again, as the value of N increases, 
// you’ll have to adjust the number of blocks you launch to get the job done.
// Here’s is the description of the tasks you have to run:
//----
// 1. Allocate space on the host for arrays hA, hB, and hC, refC, and then 
// populate hA and hB with random numbers between -10 and 10. Each 
// of these arrays is of size N.
// 2. Store the result of hA+hB into refC
// 3. Allocate space on the device for dA, dB, and dC
// 4. For inclusive timing, start the timing now
// 5. Copy content of hA and hB into dA and dB, respectively.
// 6. For exclusive timing, only start the timing now
// 7. Invoke kernel that sums up the two arrays
// 8. For exclusive timing, stop the timing now
// 9. Copy the content of dC back into hC
// 10.For inclusive timing, stop the timing now
// 11.Report the amount of time required to complete the job
// 12.Confirm that the numbers in refC and hC are identical within 1E-12



//Adds the vectors together
__global__ void add_Vector( float *hA, float *hB, float *hC)
{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		
		hC[i] = hA[i] + hB[i];
			
}

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>




int N = (int) pow(2.0,10);


//Threads that are going to be tested
int threads_per_block_test[2] = {32, 1024};


//Creates random numbers
float fRand()
{
	//Chooses if the number is negative
	float options[2] = {-1, 1};
	int op = rand()%2;
	float max = (float) rand()/RAND_MAX * 10;
	float cap = rand()%12;

	float precision = 0;

	int i;
	for(i = 0; i < cap; i ++)
	{

		precision += (float)(rand()/RAND_MAX) * (1.0/pow(10.0,i+1));
	}
	return (max + precision) * options[op];

}


int main(void)
{	
	int j, k;


	int threads_per_block;

	//Goes through Test 1(32 Threads) and Test 2(1024 Threads)
	for(j = 0; j < 2; j++)
	{	
		threads_per_block = threads_per_block_test[j];

		printf("Threads: %d\n", threads_per_block);

		for(k = 0; k <= 10; k++)
		{		

			N = pow(2.0, k +10);
			printf("\tValue of N: %d\n\t", N);
			//Host variables
			float *hA;
			float *hB;
			float *hC;
			float *refC;

			//Device variables
			float *dA, *dB, *dC;

			//Size of float
			const int size = N * sizeof(float);


			//Inclusive time
			float incTime;

			//Exclusive time
			float excTime;

			//Initialize the cuda events
			hipEvent_t incTimeStart, incTimeStop;
			hipEvent_t excTimeStart, excTimeStop;


			//Create the timing events
			hipEventCreate(&incTimeStart);
			hipEventCreate(&incTimeStop);
			hipEventCreate(&excTimeStart);
			hipEventCreate(&excTimeStop);

			//Allocating space on the host
			hA = (float *)malloc(size);
			hB = (float *)malloc(size);
			hC = (float *)malloc(size);
			refC = (float *)malloc(size);

		   	//Loop for assigning random numbers
			int i;
			for(i = 0; i < N; i++)
			{

				hA[i] = fRand();
				hB[i] = fRand();
			
			
			}

			//Sequential sum of arrays A + B
			for( i = 0; i < N; i ++)
			{
				refC[i] = hA[i] + hB[i];

			}

			//Allocation of space on the device
			hipMalloc((void**)&dA, size);
			hipMalloc((void**)&dB, size);
			hipMalloc((void**)&dC, size);


			//Inclusive Time Start
			hipEventRecord(incTimeStart, 0);

			//Copy content of hA and hB into dA and dB
			hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
			hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

			//Exclusive Time Start
			hipEventRecord(excTimeStart,0);

			//Call the kernel (add)

			add_Vector<<<N/threads_per_block, threads_per_block>>>(dA,dB, dC);

			//Exclusive Time Stop
			hipEventRecord(excTimeStop, 0);

			//Synchronize time
			hipEventSynchronize(excTimeStop);

			//Elapsed Exclusive Time
			hipEventElapsedTime(&excTime, excTimeStart, excTimeStop);


			//Copy the result back to hC (array C)
			hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

			//Inclusive Time Stop
			hipEventRecord(incTimeStop, 0);

			//Elapsed Inclusive Time
			hipEventSynchronize(incTimeStop);
			hipEventElapsedTime(&incTime, incTimeStart, incTimeStop);

			printf("\tExclusive Time was: %f\n\t\tInclusive Time was: %f\n\t\tTotal Time: %f\n", excTime, incTime, excTime + incTime);

			int k;
			//Check if the results match (refC == hC)
			for(k = 0; k < N; k++)
			{

				if(abs(refC[k] - hC[k]) > (float)(1/pow(10.0, 12)))
				{	
					///printf("%d == %d\n", ref[i], hC[i]);
					printf("The results are not identical\n");
				}
			}




			//Free the space
			free(hA);
			free(hB);
			free(hC);
			free(refC);
			hipFree(dA);
			hipFree(dB);
			hipFree(dC);
			hipEventDestroy(incTimeStart);
			hipEventDestroy(incTimeStop);
			hipEventDestroy(excTimeStart);
			hipEventDestroy(excTimeStop);

		}
	}
	return 0;


}

